#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello() {
    printf('Hello from GPU thread %d!\n', threadIdx.x);
}

int main() {
    printf('=== REAL CUDA TEST ===\n');
    
    int count;
    hipGetDeviceCount(&count);
    printf('CUDA devices: %d\n', count);
    
    if (count > 0) {
        hello<<<1, 3>>>();
        hipDeviceSynchronize();
        printf('✅ CUDA working!\n');
    }
    
    return 0;
}
