#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    printf("=== REAL CUDA GPU TEST ===\n");
    
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
        return -1;
    }
    
    printf("CUDA Devices: %d\n", deviceCount);
    
    if (deviceCount > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("GPU: %s\n", prop.name);
        printf("Memory: %.2f GB\n", prop.totalGlobalMem / 1e9);
        printf("CUDA Cores: %d\n", prop.multiProcessorCount * 128);
        printf("Clock Rate: %.2f GHz\n", prop.clockRate / 1e6);
        printf("✅ CUDA IS WORKING!\n");
        return 0;
    } else {
        printf("❌ No CUDA devices found\n");
        return -1;
    }
}
