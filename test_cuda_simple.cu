#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    printf("Device count: %d\n", deviceCount);
    
    if (deviceCount > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("Device 0: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    }
    
    return 0;
}
